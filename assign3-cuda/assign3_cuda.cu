#include "hip/hip_runtime.h"
#include "ctools.h"

__global__ void gaussianNormalization(float *A, int N, int pivot) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float tmp = A[pivot*N+tid];
    if(tid > pivot && tid < N) {
        tmp /= A[pivot*N+pivot];
    }
    A[pivot*N+tid] = tmp;
}

__global__ void setToOne(float* A, int N, int pivot) {
    A[pivot*N+pivot] = 1.0;
}

__global__ void gaussianElimination(float *A, int N, int pivot) {
    int rowId = blockIdx.x * blockDim.x + threadIdx.x;
    if(rowId > pivot && rowId < N) {
        float scale = A[rowId*N+pivot];
        for(int i=pivot+1; i<N; i++) {
            A[rowId*N+i] -= scale*A[pivot*N+i];
        }
        A[rowId*N+pivot] = 0;
    }
}

__global__ void gaussianElimination2(float *A, int N, int pivot) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float scale = A[row*N+pivot];
    if(row > pivot && row < N) {
        if(col>=pivot && col<N) {
            A[row*N+col] -= scale*A[pivot*N+col];
        }
    }
}

int main(int argc, char *argv[]) {
    int N = atoi(argv[1]);
    struct timeval begin, end;
    gettimeofday(&begin, 0);
    int NUM_THREADS = atoi(argv[2]);
    int NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;
    dim3 threads(NUM_THREADS, NUM_THREADS);
    dim3 blocks(NUM_BLOCKS, NUM_BLOCKS);
    size_t bytes = N * N * sizeof(float);

    float *h_a;
    hipHostMalloc(&h_a, bytes);

    h_a = generateMatrix(N);

    float *d_a;
    hipMalloc(&d_a, bytes);
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);

    for(int i=0; i<N; i++) {
        gaussianNormalization<<<NUM_BLOCKS, NUM_THREADS>>>(d_a, N, i);
        hipDeviceSynchronize();
        setToOne<<<1, 1>>>(d_a, N, i);
        gaussianElimination2<<<blocks, threads>>>(d_a, N, i);
        hipDeviceSynchronize();
    }

    gettimeofday(&end, 0);
    double duration = (end.tv_sec - begin.tv_sec) + (end.tv_usec - begin.tv_usec) * 1e-6;
    printf("Cuda Gaussian Elimination takes %lf seconds\n", duration);
    hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost);

    //printMatrix(h_a, N, N);

    hipFree(d_a);

    return 0;
}